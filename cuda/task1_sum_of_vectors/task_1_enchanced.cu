
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "assert.h"
#include "math.h"

#define N 100000
#define BLOCK_SIZE 128
#define GRID_SIZE 128
#define MAX_ERR 1e-6

__global__ void add(int *a, int *b, int *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (tid < n) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main() {
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	
	// Allocate host memory
    a   = (int*)malloc(sizeof(int) * N);
    b   = (int*)malloc(sizeof(int) * N);
    c = (int*)malloc(sizeof(int) * N);
	
    // Initialize host arrays
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }
	
	// Allocate device memory
	hipMalloc((void**)&d_a, sizeof(int) * N);
    hipMalloc((void**)&d_b, sizeof(int) * N);
    hipMalloc((void**)&d_c, sizeof(int) * N);
	
	// Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    add<<<BLOCK_SIZE, GRID_SIZE>>>(d_a, d_b, d_c, N);
	
	// Transfer data back to host memory
    hipMemcpy(c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++) {
        assert(abs(c[i] - a[i] - b[i]) < MAX_ERR);
    }
	
	printf("PASSED\n");
	
	// Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
