
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "assert.h"
#include "math.h"

#define N 100
#define MAX_ERR 1e-6

__global__ void add(int *a, int *b, int *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N], b[N], c[N];
	int *d_a, *d_b, *d_c;

    // Initialize host arrays
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }
	
	// Allocate device memory
	hipMalloc((void**)&d_a, sizeof(int) * N);
    hipMalloc((void**)&d_b, sizeof(int) * N);
    hipMalloc((void**)&d_c, sizeof(int) * N);
	
	// Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
    

    /*
        TEST 1
    */
    // Executing kernel 
    add<<<1,N>>>(d_a, d_b, d_c, N);
	
	// Transfer data back to host memory
    hipMemcpy(c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++) {
        assert(abs(c[i] - a[i] - b[i]) < MAX_ERR);
    }
	
	printf("<1, N> case PASSED\n");
	
	// Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);


    /*
        TEST 2
    */
    // Executing kernel 
    add<<<N,1>>>(d_a, d_b, d_c, N);
	
	// Transfer data back to host memory
    hipMemcpy(c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(abs(c[i] - a[i] - b[i]) < MAX_ERR);
    }
	
	printf("<N, 1> case PASSED\n");
	
	// Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
